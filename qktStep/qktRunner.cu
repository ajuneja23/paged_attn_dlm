#include "hip/hip_runtime.h"
#include "qktRunner.cuh"


template <int qkv_dim>
__global__ void qkt_kernel_wrapper(__half* q, __half* k, float* qkt, int b_r, int b_c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;
    calcQKT<qkv_dim>(q, k, qkt, lane_id, warp_id, b_c, b_r);
    printf("Debug: qkt last element %f\n", qkt[b_r * b_c - 1]);
}


int main(int argc, char *argv[]) {
  constexpr int qkv_dim = 64;
  constexpr int b_r = 32;
  constexpr int b_c = 32;
  std::mt19937 gen(42);
  std::uniform_real_distribution<float> dis(0.0f, 1.0f);
  __half *h_q = new __half[b_r * qkv_dim];
  float *cpu_q = new float[b_r * qkv_dim];
  float *cpu_k = new float[b_c * qkv_dim];
  __half *h_k = new __half[b_c * qkv_dim];
  float *h_qkt = new float[b_r * b_c];
  for (int i = 0; i < b_r * qkv_dim; i++) {
    cpu_q[i] = dis(gen);
    h_q[i] = __float2half(cpu_q[i]);
  }
  for (int i = 0; i < b_c * qkv_dim; i++) {
    cpu_k[i] = dis(gen);
    h_k[i] = __float2half(cpu_k[i]);
  }
  for (int i = 0; i < b_r * b_c; i++) {
    h_qkt[i] = 0.0f;
  }
  __half *d_q;
  __half *d_k;
  float *d_qkt;
  hipMalloc(&d_q, b_r * qkv_dim * sizeof(__half));
  hipMalloc(&d_k, b_c * qkv_dim * sizeof(__half));
  hipMalloc(&d_qkt, b_r * b_c * sizeof(float));
  hipMemcpy(d_q, h_q, b_r * qkv_dim * sizeof(__half), hipMemcpyHostToDevice);
  hipMemcpy(d_k, h_k, b_c * qkv_dim * sizeof(__half), hipMemcpyHostToDevice);
  hipMemcpy(d_qkt, h_qkt, b_r * b_c * sizeof(float), hipMemcpyHostToDevice);
  dim3 numBlocks(1);
  dim3 threadsPerBlock(WARP_SIZE * 4);

  qkt_kernel_wrapper<qkv_dim><<<numBlocks, threadsPerBlock>>>(d_q, d_k, d_qkt, b_r, b_c);
  hipDeviceSynchronize();
  hipMemcpy(h_qkt, d_qkt, b_r * b_c * sizeof(float), hipMemcpyDeviceToHost);
  // CPU TEST
  float *cpu_qkt = new float[b_r * b_c];
  naive_qkt<qkv_dim>(cpu_q, cpu_k, cpu_qkt, b_r, b_c);
  float allowedError = 1e-1;
  std::cout << "last element from gpu qkt uncasted: " << h_qkt[b_r * b_c - 1] << std::endl;
//   for (int i = 0; i < b_r; i++) {
//     for (int j = 0; j < b_c; j++) {
//       float diff = fabs(h_qkt[i * b_c + j] - cpu_qkt[i * b_c + j]);
//       if (diff > allowedError) {
//         std::cout << "Error at (" << i << "," << j << ")" << std::endl;
//         std::cout << "Device value: " << h_qkt[i * b_c + j] << std::endl;
//         std::cout << "CPU value: " << cpu_qkt[i * b_c + j] << std::endl;
//         std::cout << "Difference: " << diff << std::endl;
//       }
//     }
//   }
  return 0;
}