#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>
#include "fa1_forward.cuh"



template <typename T1, typename T2,int b_c, int b_r, int qkv_dim>
__device__ void calcQKT(T1* shared_q, T1* shared_k, T2* shared_qkt,int seq_len, int laneid,int warpid) {
    int req_x_tiles=ceil(b_c/TILE_Y_SIZE);
    int req_y_tiles=ceil(b_r/TILE_X_SIZE);
    int req_tiles=req_x_tiles*req_y_tiles;//# of tiles in full qk^t block output
    for (int i=warpid;i<req_tiles;i+=WARPS_PER_BLOCK) {

        int x_idx=(i)%req_x_tiles;
        int y_idx=(i)/req_x_tiles;
        int output_tile_uleft[2]={y_idx*TILE_Y_SIZE,x_idx*TILE_X_SIZE};//upper left's row, col
        T2 rC[4]={0,0,0,0};
        for (int j=0;j<qkv_dim/SHARED_Q_K_DIM;j++) {
            int q_uleft[2]={output_tile_uleft[0],j*SHARED_Q_K_DIM};
            int k_uleft[2]={output_tile_uleft[1],j*SHARED_Q_K_DIM};//storing transpose directly, row wise traversal for both Q, K tile
            T1 q_elements[8]={
                shared_q[(q_uleft[0]+laneid/4)*qkv_dim+q_uleft[1]+2*(laneid%4)],
                shared_q[(q_uleft[0]+laneid/4)*qkv_dim+q_uleft[1]+2*(laneid%4)+1],
                shared_q[(q_uleft[0]+laneid/4+8)*qkv_dim+q_uleft[1]+2*(laneid%4)],
                shared_q[(q_uleft[0]+laneid/4+8)*qkv_dim+q_uleft[1]+2*(laneid%4)+1],
                shared_q[(q_uleft[0]+laneid/4)*qkv_dim+q_uleft[1]+8+2*(laneid%4)],
                shared_q[(q_uleft[0]+laneid/4)*qkv_dim+q_uleft[1]+8+2*(laneid%4)+1],
                shared_q[(q_uleft[0]+laneid/4+8)*qkv_dim+q_uleft[1]+8+2*(laneid%4)],
                shared_q[(q_uleft[0]+laneid/4+8)*qkv_dim+q_uleft[1]+8+2*(laneid%4)+1]
            };//thank you to https://veitner.bearblog.dev/ for making the register loading a lot easier
                T1 k_elements[4]={
                    shared_k[(k_uleft[0]+laneid/4)*qkv_dim+k_uleft[1]+2*(laneid%4)],
                    shared_k[(k_uleft[0]+laneid/4)*qkv_dim+k_uleft[1]+2*(laneid%4)+1],
                    shared_k[(k_uleft[0]+laneid/4)*qkv_dim+k_uleft[1]+2*(laneid%4)+8],
                    shared_k[(k_uleft[0]+laneid/4)*qkv_dim+k_uleft[1]+2*(laneid%4)+9]//danger
                };
                //use ptx instruction!
                    asm("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"//just handling the f32 accum f16 mat A,B pattern for now
                "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
                : "=f"(rC[0]), "=f"(rC[1]), "=f"(rC[2]), "=f"(rC[3])
                : "r"(q_elements[0]), "r"(q_elements[1]), "r"(q_elements[2]), "r"(q_elements[3]), "r"(k_elements[0]), "r"(k_elements[1]),
                    "f"(rC[0]), "f"(rC[1]), "f"(rC[2]), "f"(rC[3]));
    }
    //store to smem
    shared_qkt[(output_tile_uleft[0]+laneid/4)*b_c+output_tile_uleft[1]+2*(laneid%4)]=rC[0];
    shared_qkt[(output_tile_uleft[0]+laneid/4)*b_c+output_tile_uleft[1]+2*(laneid%4)+1]=rC[1];
    shared_qkt[(output_tile_uleft[0]+laneid/4+8)*b_c+output_tile_uleft[1]+2*(laneid%4)]=rC[2];
    shared_qkt[(output_tile_uleft[0]+laneid/4+8)*b_c+output_tile_uleft[1]+2*(laneid%4)+1]=rC[3];
    }
}


template<typename T1, typename T2, int b_c, int b_r>
__device__ void reductionStep(T2* shared_qkt, T2* maxValues, T2* sumValues, T1* shared_v,T2* output, T2* intermediateRowMaxes, T2* intermediatePV, T1* casted_qkt, int warpid, int laneid) {
    //calculate maxValues, P_{ij} matrix, and l_ij values. split work for each row across warps

    for (int i=warpid;i<b_r;i+=WARPS_PER_BLOCK) {
        T2 m_ijProposal=-INFINITY;
        for (int j=laneid;j<b_c;j+=WARP_SIZE) {
            T2 m_ijProposal=max(m_ijProposal,shared_qkt[i][j]);
        }
        for (int offset=WARP_SIZE/2;offset>0;offset>>=1) {
            m_ijProposal=max(m_ijProposal,__shfl_down_sync(0xFFFFFFFF,m_ijProposal,offset));
        }
        if (laneid == 0) {
            maxValues[i]=max(maxValues[i],m_ijProposal);
        }
        m_ijProposal=__shfl_sync(0xFFFFFFFF,m_ijProposal,0);
        T2 runningSum=0;
        for (int j=laneid;j<b_c;j+=WARP_SIZE) {
            shared_qkt[i][j]-=m_ijProposal;
            shared_qkt[i][j]=exp(shared_qkt[i][j]);
            runningSum+=shared_qkt[i][j];
        }
        for (int offset=WARP_SIZE/2;offset>0;offset>>=1) {
            runningSum+=__shfl_down_sync(0xFFFFFFFF,runningSum,offset);//l_{ij} calculation
        }
        runningSum=__shfl_sync(0xFFFFFFFF,runningSum,0);
        T2 curMax=maxValues[i];
        T2 curRunningSum=sumValues[i];//m_i
        T2 l_inew=exp(curMax-max(curMax,m_ijProposal))*curRunningSum+exp(m_ijProposal-max(curMax,m_ijProposal))*runningSum;//l_i^{new} 
        if (laneid == 0) {
            intermediateRowMaxes[i]=m_ijProposal;
        }
        //update O_i
        for (int j=laneid;j<qkv_dim;j+=WARP_SIZE) {
            output[i][j]=(curRunningSum/l_inew)*exp(curMax-max(curMax,m_ijProposal))*output[i][j];
        }
        sumValues[i]=l_inew;
        __syncthreads();
    }
    //cast qkt to T1
    for (int i=tid;i<b_r*b_c;i+=WARP_SIZE*WARPS_PER_BLOCK) {
        casted_qkt[i/b_c][i%b_c]=shared_qkt[i/b_c][i%b_c];
    }
    __syncthreads();
    //handle p_{ij} by v_j multiplication. p_{ij} is in casted_qkt as a b_r x b_c(16x16 tiling). v_j is shared_v as a b_c x qkv_dim (16x8 tiling) 
    int req_x_tiles=ceil(qkv_dim/TILE_X_SIZE);
    int req_y_tiles=ceil(b_c/TILE_Y_SIZE);
    int req_tiles=req_x_tiles*req_y_tiles;
    for (int i=warpid;i<req_tiles;i+=WARPS_PER_BLOCK) {
        T2 rC[4]={0,0,0,0};
        int output_u_left[2]={(i)/req_x_tiles*TILE_Y_SIZE,(i)%req_x_tiles*TILE_X_SIZE};//split output tile work across warps 
        for (int j=0;j<(b_c/SHARED_Q_K_DIM);j++) {
            int p_u_left[2]={output_u_left[0],j*SHARED_Q_K_DIM};
            int v_u_left[2]={j*SHARED_Q_K_DIM,output_u_left[1]};
            T1 p_elements[8]={
                casted_qkt[(p_u_left[0]+laneid/4)*b_cp_u_left[1]+2*(laneid%4)],
                casted_qkt[(p_u_left[0]+laneid/4)*b_c+p_u_left[1]+2*(laneid%4)+1],
                casted_qkt[(p_u_left[0]+laneid/4+8)*b_c+p_u_left[1]+2*(laneid%4)],
                casted_qkt[(p_u_left[0]+laneid/4+8)*b_c+p_u_left[1]+2*(laneid%4)+1],
                casted_qkt[(p_u_left[0]+laneid/4)*b_c+p_u_left[1]+8+2*(laneid%4)],
                casted_qkt[(p_u_left[0]+laneid/4+8)*b_c+p_u_left[1]+8+2*(laneid%4)+1],
                casted_qkt[(p_u_left[0]+laneid/4)*b_c+p_u_left[1]+8+2*(laneid%4)],
                casted_qkt[(p_u_left[0]+laneid/4+8)*b_c+p_u_left[1]+8+2*(laneid%4)+1]
            };
            T1 v_elements[4]={
                shared_v[(v_u_left[0]+2*(laneid%4))*qkv_dim+v_u_left[1]+laneid/4],
                shared_v[(v_u_left[0]+2*(laneid%4)+1)*qkv_dim+v_u_left[1]+laneid/4],
                shared_v[(v_u_left[0]+2*(laneid%4)+8)*qkv_dim+v_u_left[1]+laneid/4],
                shared_v[(v_u_left[0]+2*(laneid%4)+9)*qkv_dim+v_u_left[1]+laneid/4]
            };
            //use ptx instruction!
            asm("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"//just handling the f32 accum f16 mat A,B pattern for now
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(rC[0]), "=f"(rC[1]), "=f"(rC[2]), "=f"(rC[3])
        : "r"(casted_qkt[0]), "r"(casted_qkt[1]), "r"(casted_qkt[2]), "r"(casted_qkt[3]), "r"(v_elements[0]), "r"(v_elements[1]),
            "f"(rC[0]), "f"(rC[1]), "f"(rC[2]), "f"(rC[3]));
        }
        intermediatePV[(output_u_left[0]+laneid/4)*qkv_dim+output_u_left[1]+2*(laneid%4)]=rC[0];
        intermediatePV[(output_u_left[0]+laneid/4)*qkv_dim+output_u_left[1]+2*(laneid%4)+1]=rC[1];
        intermediatePV[(output_u_left[0]+laneid/4+8)*qkv_dim+output_u_left[1]+2*(laneid%4)]=rC[2];
        intermediatePV[(output_u_left[0]+laneid/4+8)*qkv_dim+output_u_left[1]+2*(laneid%4)+1]=rC[3];
    }
    __syncthreads();
    //final O_i update
    for (int i=warpid;i<b_r;i+=WARPS_PER_BLOCK) {
        T2 coefficient=exp(intermediateRowMaxes[i]-maxValues[i])/sumValues[i];
        for (int j=laneid;j<qkv_dim;j+=WARP_SIZE) {
            output[i][j]+=coefficient*intermediatePV[i][j];
        }

    }



}



//parallelize on heads first
template<typename T1, typename T2,int qkv_dim, int num_heads>
__global__ void fa1_fwd(T1* q, T1* k, T1* v, T2* maxValues, T2* sumValues, T2* output,int seq_len)
{//q layout is (qkv_dim,seq_len,num_heads): (1, qkv_dim,qkv_dim*seq_len). same for k,v 
    int tid=threadIdx.y*blockDim.x+threadIdx.x;
    int bid=blockIdx.y*gridDim.x+blockIdx.x;
    int b_c=seq_len/(4*qkv_dim);
    int b_r=min(b_c,qkv_dim);
    extern __shared__ T1 shared_q[b_r][qkv_dim];
    extern __shared__ T1 shared_k[b_c][qkv_dim];
    extern __shared__ T1 shared_v[b_c][qkv_dim]; 
    extern __shared__ T2 shared_maxValues[b_r];
    extern __shared__ T2 shared_sumValues[b_r];
    extern __shared__ T2 shared_output[b_r][qkv_dim];
    extern __shared__ T2 shared_qkt[b_r][b_c];
    extern __shared__ T2 shared_intermediateRowMaxes[b_r];
    extern __shared__ T1 casted_qkt[b_r][b_c];
    extern __shared__ T2 shared_intermediatePV[b_r][qkv_dim];
    int warpid=tid/WARP_SIZE;
    int laneid=tid%WARP_SIZE;

    int head_id=bid;
    if (bid < num_heads) {//bid=head_id
        int head_prefix=head_id*seq_len*qkv_dim;
        int b_c=seq_len/(4*qkv_dim);//split k,v into tiles of this size on seq_len dim 
        int b_r=min(b_c,qkv_dim);//split q into tiles of this on seq_len dim
        int t_c=ceil(seq_len/b_c);
        int t_r=ceil(seq_len/b_r);
        for (int j=0;j<t_c;j++) {//load in qkv_dim*b_c elements
            int elementsToLoad=b_c*qkv_dim;
            int seq_prefix=j*b_c*qkv_dim;
            for (int k=0;k<elementsToLoad;k+=(WARP_SIZE*WARPS_PER_BLOCK)) {
                if (k+tid<elementsToLoad) {
                    shared_k[(k+tid)/qkv_dim][(k+tid)%qkv_dim]=k[head_prefix+seq_prefix+k+tid];
                    shared_v[(k+tid)/qkv_dim][(k+tid)%qkv_dim]=v[head_prefix+seq_prefix+k+tid];
                }//split k pattern
            }
            __syncthreads();
            for (int i=0;i<t_r;i++) {
                int q_prefix=i*b_r*qkv_dim; 
                int elementsToLoad=b_r*qkv_dim;
                for (int k=0;k<elementsToLoad;k+=(WARP_SIZE*WARPS_PER_BLOCK)) {
                    if (k+tid<elementsToLoad) {
                        shared_q[(k+tid)/qkv_dim][(k+tid)%qkv_dim]=q[head_prefix+q_prefix+k+tid];
                    }

                }
            }
            //load in maxValues, sumValues

            __syncthreads();
            calcQKT<T1,T2,b_c,qkv_dim>(shared_q,shared_k,shared_qkt,seq_len,laneid,warpid);
            __syncthreads(); 
            //load in all required sram utils from dram 
            //first half of warps load in maxValues, second half load in sumValues
            if (warpid < WARPS_PER_BLOCK/2) {
                for(int k=tid;k<b_r;k+=(WARP_SIZE*WARPS_PER_BLOCK/2)) {
                    shared_maxValues[k]=maxValues[i*b_r+k];
                }
            } else {
                for (int k=tid-(WARP_SIZE*WARPS_PER_BLOCK/2);k<b_r;k+=(WARP_SIZE*WARPS_PER_BLOCK/2)) {
                    shared_sumValues[k]=sumValues[i*b_r+k];
            }
            }
            //collaborate on O block loading
            for (int k=tid;k<b_r*qkv_dim;k+=(WARP_SIZE*WARPS_PER_BLOCK)) {
                shared_output[k/qkv_dim][k%qkv_dim]=output[head_prefix+(b_r*i+k/qkv_dim)*qkv_dim+(k%qkv_dim)];
            }
            __syncthreads();
            reductionStep<T1,T2,b_c,qkv_dim>(shared_qkt,shared_maxValues,shared_sumValues,shared_v,shared_output,shared_intermediateRowMaxes,shared_intermediatePV,casted_qkt,warpid,laneid);
            __syncthreads();
            //write output to DRAM
            if (warpid < WARPS_PER_BLOCK/2) {
                for(int k=tid;k<b_r;k+=(WARP_SIZE*WARPS_PER_BLOCK/2)) {
                    maxValues[i*b_r+k]=shared_maxValues[k];
                }
            } else {
                for (int k=tid-(WARP_SIZE*WARPS_PER_BLOCK/2);k<b_r;k+=(WARP_SIZE*WARPS_PER_BLOCK/2)) {
                    sumValues[i*b_r+k]=shared_sumValues[k];
            }
            }
            //collaborate on O block loading
            for (int k=tid;k<b_r*qkv_dim;k+=(WARP_SIZE*WARPS_PER_BLOCK)) {
                output[head_prefix+(b_r*i+k/qkv_dim)*qkv_dim+(k%qkv_dim)]=shared_output[k/qkv_dim][k%qkv_dim];
            }
        }
    }
}


__host__ void fa1_fwd_wrapper() {

    int seq_len = 1024;
    int qkv_dim = 1024;
    int num_heads = 16;
    float* d_q;
    float* d_k;
    float* d_v;
    float* d_maxValues;
    float* d_sumValues;
    float* d_output;

    hipMalloc(&d_q, num_heads * seq_len * qkv_dim * sizeof(float));
    hipMalloc(&d_k, num_heads * seq_len * qkv_dim * sizeof(float));
    hipMalloc(&d_v, num_heads * seq_len * qkv_dim * sizeof(float));
    hipMalloc(&d_maxValues, num_heads * seq_len * sizeof(float));
    hipMalloc(&d_sumValues, num_heads * seq_len * sizeof(float));
    hipMalloc(&d_output, num_heads * seq_len * qkv_dim * sizeof(float));
    float* h_q = new float[num_heads * seq_len * qkv_dim];
    float* h_k = new float[num_heads * seq_len * qkv_dim];
    float* h_v = new float[num_heads * seq_len * qkv_dim];
    for (int i = 0; i < num_heads * seq_len * qkv_dim; ++i) {
        h_q[i] = static_cast<float>(rand()) / RAND_MAX;
        h_k[i] = static_cast<float>(rand()) / RAND_MAX;
        h_v[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    float* h_maxValues = new float[num_heads * seq_len];
    float* h_sumValues = new float[num_heads * seq_len];
    for (int i = 0; i < num_heads * seq_len; ++i) {
        h_maxValues[i] = -std::numeric_limits<float>::infinity();
        h_sumValues[i] = 0.0f;
    }
    hipMemcpy(d_q, h_q, num_heads * seq_len * qkv_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, num_heads * seq_len * qkv_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, num_heads * seq_len * qkv_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_maxValues, h_maxValues, num_heads * seq_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sumValues, h_sumValues, num_heads * seq_len * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8,16);
    dim3 numBlocks((seq_len + threadsPerBlock.x - 1) / threadsPerBlock.x, num_heads);
    auto kernel_call=fa1_fwd<__half, float, 1024, 16>;
    kernel_call <<<numBlocks, threadsPerBlock>>>(
        d_q, 
        d_k, 
        d_v, 
        d_maxValues, 
        d_sumValues, 
        d_output, 
        seq_len
    );

    // Copy the result back to host
    float* h_output = new float[num_heads * seq_len * qkv_dim];
    hipMemcpy(h_output, d_output, num_heads * seq_len * qkv_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < num_heads * seq_len * qkv_dim; ++i) {
        std::cout << "output[" << i << "]: " << h_output[i] << std::endl;
    }
    delete[] h_q;
    delete[] h_k;
    delete[] h_v;
    delete[] h_maxValues;
    delete[] h_sumValues;
    delete[] h_output;
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_maxValues);
    hipFree(d_sumValues);
    hipFree(d_output);
}